
#include <hip/hip_runtime.h>
/*******************************************************************************
 * serveral useful gpu functions will be defined in this file to facilitate
 * the extension scheme 
 ******************************************************************************/

typedef struct
{
	double sR;
	double sL;
} double_eno_derivative;

__device__ inline
double max2(double x, double y)
{
	return (x<y) ? y : x;
}

__device__ inline
double min2(double x, double y)
{
	return (x<y) ? x : y;
}

__device__ inline
double min_mod(double x, double y)
{
	return (x*y<0) ? 0.0 : (fabs(x)<fabs(y) ? x : y);
}


__device__ inline
double sign(double x)
{
	return (x>0) ? 1.0 : -1.0;
}


// convert subindex to linear index
// periodic boundary conditions are assumed
__device__ inline
int sub2ind(int row_idx, int col_idx, int pge_idx, int rows, int cols, int pges)
{	
	int row_idxn = min2(rows-1, max2(0, row_idx));
	int col_idxn = min2(cols-1, max2(0, col_idx));
	int pge_idxn = min2(pges-1, max2(0, pge_idx));

	int ind = pge_idxn * rows * cols + col_idxn * rows + row_idxn;

	return ind;
}

/****************************************************************************** 
 * calculate Eno derivatives at node v0: [v4,v1,v0,v2,v3]
 * without boundary correction
 ******************************************************************************/
__device__ inline
double_eno_derivative eno_derivative_field( double v4, double v1, double v0, double v2, double v3, double ds)
{
	double p2m;
	double_eno_derivative eno_d;

	double p2 = v1 - 2.0 * v0 + v2;

	double p2r = v0 - 2.0 * v2 + v3;
	p2m = 0.5 * min_mod(p2, p2r) / pow(ds, 2);
	eno_d.sR = (v2 - v0) / ds - ds * p2m;

	double p2l = v0 - 2.0 * v1 + v4;
	p2m = 0.5 * min_mod(p2, p2l) / pow(ds, 2);
	eno_d.sL = (v0 - v1) / ds + ds * p2m;

	return eno_d;

}

/*******************************************************************************
 * calculate upwind normal with ENO scheme
 *******************************************************************************/                                                                                 
__device__ inline
double upwind_normal_point( double v4, double v1, double v0, double v2, double v3, double pr, double pl, double ds)
{
	double p2m;

	double p2 = v1 - 2.0 * v0 + v2;

	double p2r = v0 - 2.0 * v2 + v3;
	p2m = 0.5 * min_mod(p2, p2r) / pow(ds, 2);
	double vr = (pr==ds) ? v2 : 0;
	double sR = (vr - v0) / pr - pr * p2m;

	double p2l = v0 - 2.0 * v1 + v4;
	p2m = 0.5 * min_mod(p2, p2l) / pow(ds, 2);
	double vl = (pl==ds) ? v1 : 0;
	double sL = (v0 - vl) / pl + pl * p2m;

	return (fabs(vr) < fabs(vl)) ? sR : sL;
}

// calculate the upwind normal
__global__
void upwind_normal(double * nx, double * ny, double * nz, double const * lsf, double const * xpr, double const * xpl, double const * ypf, double const * ypb, double const * zpu, double const * zpd, int rows, int cols, int pges, double dx, double dy, double dz, int num_ele)
{

	int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int col_idx = blockIdx.y * blockDim.y + threadIdx.y;
	int pge_idx = blockIdx.z * blockDim.z + threadIdx.z;

	if(row_idx >= rows || col_idx >= cols || pge_idx >= pges){
		return;
	}

	int ind = sub2ind(row_idx, col_idx, pge_idx, rows, cols, pges);

	int right 	= sub2ind(row_idx, col_idx+1, pge_idx, rows, cols, pges);
	int right2 	= sub2ind(row_idx, col_idx+2, pge_idx, rows, cols, pges);
	int left 	= sub2ind(row_idx, col_idx-1, pge_idx, rows, cols, pges);
	int left2 	= sub2ind(row_idx, col_idx-2, pge_idx, rows, cols, pges);

	nx[ind] = upwind_normal_point( lsf[left2], lsf[left], lsf[ind], lsf[right], lsf[right2], xpr[ind], xpl[ind], dx);

	int front 	= sub2ind(row_idx+1, col_idx, pge_idx, rows, cols, pges);	
	int front2 	= sub2ind(row_idx+2, col_idx, pge_idx, rows, cols, pges);
	int back 	= sub2ind(row_idx-1, col_idx, pge_idx, rows, cols, pges);
	int back2 	= sub2ind(row_idx-2, col_idx, pge_idx, rows, cols, pges);

	ny[ind] = upwind_normal_point( lsf[back2], lsf[back], lsf[ind], lsf[front], lsf[front2], ypf[ind], ypb[ind], dy);

	int up 		= sub2ind(row_idx, col_idx, pge_idx+1, rows, cols, pges);	
	int up2 	= sub2ind(row_idx, col_idx, pge_idx+2, rows, cols, pges);	
	int down 	= sub2ind(row_idx, col_idx, pge_idx-1, rows, cols, pges);
	int down2 	= sub2ind(row_idx, col_idx, pge_idx-2, rows, cols, pges);

	nz[ind] = upwind_normal_point( lsf[down2], lsf[down], lsf[ind], lsf[up], lsf[up2], zpu[ind], zpd[ind], dz);

}	

// calculate extend step
// now lsf represents a scalar field (not the level set function)
__global__
void extend_step(double * step, double const * lsf, bool const * boundary, double const * vx, double const * vy, double const * vz, int rows, int cols, int pges, double dx, double dy, double dz, int num_ele)
{
	int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int col_idx = blockIdx.y * blockDim.y + threadIdx.y;
	int pge_idx = blockIdx.z * blockDim.z + threadIdx.z;

	if(row_idx >= rows || col_idx >= cols || pge_idx >= pges){
		return;
	}

	int ind = sub2ind(row_idx, col_idx, pge_idx, rows, cols, pges);

	int right 	= sub2ind(row_idx, col_idx+1, pge_idx, rows, cols, pges);
	int right2 	= sub2ind(row_idx, col_idx+2, pge_idx, rows, cols, pges);
	int left 	= sub2ind(row_idx, col_idx-1, pge_idx, rows, cols, pges);
	int left2 	= sub2ind(row_idx, col_idx-2, pge_idx, rows, cols, pges);

	double_eno_derivative eno_dx = eno_derivative_field( lsf[left2], lsf[left], lsf[ind], lsf[right], lsf[right2], dx);
	double xR = eno_dx.sR;
	double xL = eno_dx.sL;


	int front 	= sub2ind(row_idx+1, col_idx, pge_idx, rows, cols, pges);	
	int front2 	= sub2ind(row_idx+2, col_idx, pge_idx, rows, cols, pges);
	int back 	= sub2ind(row_idx-1, col_idx, pge_idx, rows, cols, pges);
	int back2 	= sub2ind(row_idx-2, col_idx, pge_idx, rows, cols, pges);

	double_eno_derivative eno_dy = eno_derivative_field( lsf[back2], lsf[back], lsf[ind], lsf[front], lsf[front2], dy);
	double yF = eno_dy.sR;
	double yB = eno_dy.sL;


	int up 		= sub2ind(row_idx, col_idx, pge_idx+1, rows, cols, pges);	
	int up2 	= sub2ind(row_idx, col_idx, pge_idx+2, rows, cols, pges);	
	int down 	= sub2ind(row_idx, col_idx, pge_idx-1, rows, cols, pges);
	int down2 	= sub2ind(row_idx, col_idx, pge_idx-2, rows, cols, pges);

	double_eno_derivative eno_dz = eno_derivative_field( lsf[down2], lsf[down], lsf[ind], lsf[up], lsf[up2], dz);
	double zU = eno_dz.sR;
	double zD = eno_dz.sL;

	step[ind] = min2(0,vx[ind]) * xR + max2(0,vx[ind]) * xL + 
				min2(0,vy[ind]) * yF + max2(0,vy[ind]) * yB + 
				min2(0,vz[ind]) * zU + max2(0,vz[ind]) * zD ;

	// keep boundary values fixed
	step[ind] = boundary[ind] ? 0.0 : step[ind];
}










