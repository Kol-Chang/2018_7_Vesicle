#include "hip/hip_runtime.h"
/*******************************************************************************
 * serveral useful gpu functions will be defined in this file to facilitate
 * the extension scheme 
 ******************************************************************************/
#include "shared_utilities.cuh"
#include "shared_utilities.cup"

// calculate extend step
// now lsf represents a scalar field (not the level set function)
__global__
void extend_step(double * step, double const * deltat, double const * lsf, double const * vx, double const * vy, double const * vz, double const * xpr, double const * xpl, double const * ypf, double const * ypb, double const * zpu, double const * zpd, double const * cpr, double const * cpl, double const * cpf, double const * cpb, double const * cpu, double const * cpd, int rows, int cols, int pges, double dx, double dy, double dz, int num_ele)
{
	int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int col_idx = blockIdx.y * blockDim.y + threadIdx.y;
	int pge_idx = blockIdx.z * blockDim.z + threadIdx.z;

	if(row_idx >= rows || col_idx >= cols || pge_idx >= pges){
		return;
	}

	int ind = sub2ind(row_idx, col_idx, pge_idx, rows, cols, pges);

	double epsilon = 1e-6 * dx;
	if( xpr[ind]<epsilon || xpl[ind]<epsilon ||
		ypf[ind]<epsilon || ypb[ind]<epsilon || 
	   	zpu[ind]<epsilon || zpd[ind]<epsilon ){
		step[ind] = 0;
		return;
	}// for a boundary node, do not change its value

	double p1,p2,p3,p4,p5,p6,p7;
	double r1,r2,r3,l1,l2,l3;
	double v_fore, v_back;

	p4 = lsf[ind];

	int rght1 	= sub2ind(row_idx, col_idx+1, pge_idx, rows, cols, pges);
	int rght2 	= sub2ind(row_idx, col_idx+2, pge_idx, rows, cols, pges);
	int rght3 	= sub2ind(row_idx, col_idx+3, pge_idx, rows, cols, pges);
	int left1 	= sub2ind(row_idx, col_idx-1, pge_idx, rows, cols, pges);
	int left2 	= sub2ind(row_idx, col_idx-2, pge_idx, rows, cols, pges);
	int left3 	= sub2ind(row_idx, col_idx-3, pge_idx, rows, cols, pges);

	p1 = lsf[left3];
	p2 = lsf[left2];
	p3 = lsf[left1];
	p5 = lsf[rght1];
	p6 = lsf[rght2];
	p7 = lsf[rght3];

	r1 = xpr[ind];
	r2 = xpr[rght1];
	r3 = xpr[rght2];

	l1 = xpl[ind];
	l2 = xpl[left1];
	l3 = xpl[left2];

	v_fore = cpr[ind];
	v_back = cpl[ind];

	double xR, xL;
	weno_derivative_boundary(xR,xL,p1,p2,p3,p4,p5,p6,p7,r1,r2,r3,l1,l2,l3,dx,v_fore,v_back);

	int frnt1 	= sub2ind(row_idx+1, col_idx, pge_idx, rows, cols, pges);	
	int frnt2 	= sub2ind(row_idx+2, col_idx, pge_idx, rows, cols, pges);	
	int frnt3 	= sub2ind(row_idx+3, col_idx, pge_idx, rows, cols, pges);	
	int back1 	= sub2ind(row_idx-1, col_idx, pge_idx, rows, cols, pges);
	int back2 	= sub2ind(row_idx-2, col_idx, pge_idx, rows, cols, pges);
	int back3 	= sub2ind(row_idx-3, col_idx, pge_idx, rows, cols, pges);

	p1 = lsf[back3];
	p2 = lsf[back2];
	p3 = lsf[back1];
	p5 = lsf[frnt1];
	p6 = lsf[frnt2];
	p7 = lsf[frnt3];

	r1 = ypf[ind];
	r2 = ypf[frnt1];
	r3 = ypf[frnt2];

	l1 = ypb[ind];
	l2 = ypb[back1];
	l3 = ypb[back2];

	v_fore = cpf[ind];
	v_back = cpb[ind];

	double yF, yB;
	weno_derivative_boundary(yF,yB,p1,p2,p3,p4,p5,p6,p7,r1,r2,r3,l1,l2,l3,dy,v_fore,v_back);

	int upup1	= sub2ind(row_idx, col_idx, pge_idx+1, rows, cols, pges);	
	int upup2	= sub2ind(row_idx, col_idx, pge_idx+2, rows, cols, pges);	
	int upup3	= sub2ind(row_idx, col_idx, pge_idx+3, rows, cols, pges);	
	int down1 	= sub2ind(row_idx, col_idx, pge_idx-1, rows, cols, pges);
	int down2 	= sub2ind(row_idx, col_idx, pge_idx-2, rows, cols, pges);
	int down3 	= sub2ind(row_idx, col_idx, pge_idx-3, rows, cols, pges);

	p1 = lsf[down3];
	p2 = lsf[down2];
	p3 = lsf[down1];
	p5 = lsf[upup1];
	p6 = lsf[upup2];
	p7 = lsf[upup3];

	r1 = zpu[ind];
	r2 = zpu[upup1];
	r3 = zpu[upup2];

	l1 = zpd[ind];
	l2 = zpd[down1];
	l3 = zpd[down2];

	v_fore = cpu[ind];
	v_back = cpd[ind];

	double zU, zD;
	weno_derivative_boundary(zU,zD,p1,p2,p3,p4,p5,p6,p7,r1,r2,r3,l1,l2,l3,dz,v_fore,v_back);
	
	step[ind] = (min2(0.0,vx[ind]) * xR + max2(0.0,vx[ind]) * xL + 
				 min2(0.0,vy[ind]) * yF + max2(0.0,vy[ind]) * yB + 
				 min2(0.0,vz[ind]) * zU + max2(0.0,vz[ind]) * zD ) * deltat[ind];

}








