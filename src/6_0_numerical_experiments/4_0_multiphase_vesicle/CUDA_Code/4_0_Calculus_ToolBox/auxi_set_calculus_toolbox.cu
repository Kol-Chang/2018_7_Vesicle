#include "hip/hip_runtime.h"
/*******************************************************************************
 * serveral useful gpu functions will be defined in this file to facilitate
 * the set calculus toolbox scheme, i.e., to calculate gradients,normal vectors,
 * curvatures, Heaviside function and Dirac_Delta function 
 ******************************************************************************/
#include "shared_utilities.cuh"
#include "shared_utilities.cup"


__global__
void auxi_set_calculus_toolbox(double * Ax, double * Ay, double * Az, double * AGradMag, double * ACx, double * ACy, double * ACz, double * ANormCrossAF, double * Tx, double * Ty, double * Tz, double * Anx, double * Any, double * Anz, double * Axx, double * Ayy, double * Azz, double * Axy, double * Ayz, double * Azx, double * ALaplacian, double * GeodesicCurvature, double * NormalCurvature, double * GeodesicTorsion, double * BPerpendicular, double * AHeaviside, double * ADiracDelta, double const * lsf, double const * AHPrimal, double const * Fx, double const * Fy, double const * Fz, double const * FGradMag, double const * Nx, double const * Ny, double const * Nz, double const * Fxx, double const * Fyy, double const * Fzz, double const * Fxy, double const * Fyz, double const * Fzx, int rows, int cols, int pges, double dx, double dy, double dz, double ds, int num_ele)
{
	int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int col_idx = blockIdx.y * blockDim.y + threadIdx.y;
	int pge_idx = blockIdx.z * blockDim.z + threadIdx.z;

	if(row_idx >= rows || col_idx >= cols || pge_idx >= pges){
		return;
	}

	int ind = sub2ind(row_idx, col_idx, pge_idx, rows, cols, pges);

	int right 	= sub2ind(row_idx, col_idx+1, pge_idx, rows, cols, pges);
	int left 	= sub2ind(row_idx, col_idx-1, pge_idx, rows, cols, pges);

	int front 	= sub2ind(row_idx+1, col_idx, pge_idx, rows, cols, pges);	
	int back 	= sub2ind(row_idx-1, col_idx, pge_idx, rows, cols, pges);

	int up 		= sub2ind(row_idx, col_idx, pge_idx+1, rows, cols, pges);	
	int down 	= sub2ind(row_idx, col_idx, pge_idx-1, rows, cols, pges);

	double ax = (lsf[right] - lsf[left]) / (2*dx);
	double ay = (lsf[front] - lsf[back]) / (2*dy);
	double az = (lsf[up] - lsf[down]) / (2*dz);
	double aGradMag = norm(ax, ay, az);

	Ax[ind] = ax;
	Ay[ind] = ay;
	Az[ind] = az;
	AGradMag[ind] = aGradMag;

	double fx = Fx[ind];
	double fy = Fy[ind];
	double fz = Fz[ind];

	double Cx, Cy, Cz;
	cross_product(Cx,Cy,Cz,fx,fy,fz,ax,ay,az);
	double NormCrossAF = norm(Cx,Cy,Cz);

	ACx[ind] = Cx;
	ACy[ind] = Cy;
	ACz[ind] = Cz;
	ANormCrossAF[ind] = NormCrossAF;

	double tx = Cx / NormCrossAF;
	double ty = Cy / NormCrossAF;
	double tz = Cz / NormCrossAF;

	Tx[ind] = tx;
	Ty[ind] = ty;
	Tz[ind] = tz;

	double fNx = Nx[ind];
	double fNy = Ny[ind];
	double fNz = Nz[ind];

	double nx, ny, nz;
	cross_product(nx,ny,nz,tx,ty,tz,fNx,fNy,fNz);

	Anx[ind] = nx;
	Any[ind] = ny;
	Anz[ind] = nz;

	int front_right = sub2ind(row_idx+1, col_idx+1, pge_idx, rows, cols, pges);
	int back_left 	= sub2ind(row_idx-1, col_idx-1, pge_idx, rows, cols, pges);
	int back_right 	= sub2ind(row_idx-1, col_idx+1, pge_idx, rows, cols, pges);
	int front_left 	= sub2ind(row_idx+1, col_idx-1, pge_idx, rows, cols, pges);

	int front_up 	= sub2ind(row_idx+1, col_idx, pge_idx+1, rows, cols, pges);
	int back_down 	= sub2ind(row_idx-1, col_idx, pge_idx-1, rows, cols, pges);
	int front_down 	= sub2ind(row_idx+1, col_idx, pge_idx-1, rows, cols, pges);
	int back_up 	= sub2ind(row_idx-1, col_idx, pge_idx+1, rows, cols, pges);

	int right_up 	= sub2ind(row_idx, col_idx+1, pge_idx+1, rows, cols, pges);
	int left_down 	= sub2ind(row_idx, col_idx-1, pge_idx-1, rows, cols, pges);
	int right_down 	= sub2ind(row_idx, col_idx+1, pge_idx-1, rows, cols, pges);
	int left_up 	= sub2ind(row_idx, col_idx-1, pge_idx+1, rows, cols, pges);

	double axx = (lsf[right] - 2*lsf[ind] + lsf[left]) / (dx*dx);
	double ayy = (lsf[front] - 2*lsf[ind] + lsf[back]) / (dy*dy);
	double azz = (lsf[up] - 2*lsf[ind] + lsf[down]) / (dz*dz);
	double axy = (lsf[front_right]+lsf[back_left]-lsf[front_left]-lsf[back_right]) / (4*ds*ds);
	double ayz = (lsf[front_up]+lsf[back_down]-lsf[front_down]-lsf[back_up]) / (4*ds*ds);
	double azx = (lsf[right_up]+lsf[left_down]-lsf[right_down]-lsf[left_up]) / (4*ds*ds);
	double aLaplacian = axx + ayy + azz;

	Axx[ind] = axx;
	Ayy[ind] = ayy;
	Azz[ind] = azz;
	Axy[ind] = axy;
	Ayz[ind] = ayz;
	Azx[ind] = azx;
	ALaplacian[ind] = aLaplacian;

	// geodesic curvature
	double fxx = Fxx[ind];
	double fyy = Fyy[ind];
	double fzz = Fzz[ind];
	double fxy = Fxy[ind];
	double fyz = Fyz[ind];
	double fzx = Fzx[ind];
	double fGradMag = FGradMag[ind];

	double vx = tx*fxx + ty*fxy + tz*fzx;
	double vy = tx*fxy + ty*fyy + tz*fyz;
	double vz = tx*fzx + ty*fyz + tz*fzz;

	double w1x, w1y, w1z;
	cross_product(w1x,w1y,w1z,vx,vy,vz,ax,ay,az);

	vx = tx*axx + ty*axy + tz*azx;
	vy = tx*axy + ty*ayy + tz*ayz;
	vz = tx*azx + ty*ayz + tz*azz;

	double w2x, w2y, w2z;
	cross_product(w2x,w2y,w2z,fx,fy,fz,vx,vy,vz);

	GeodesicCurvature[ind] = ( nx*(w1x+w2x) + ny*(w1y+w2y) + nz*(w1z+w2z) ) / NormCrossAF;

	/* NormalCurvature, GeodesicTorsion, BPerpendicular */
	double Nxx = fxx / fGradMag - fx*(fxx*fx + fxy*fy + fzx*fz) / pow(fGradMag,3) ; 
	double Nyx = fxy / fGradMag - fy*(fxx*fx + fxy*fy + fzx*fz) / pow(fGradMag,3) ; 
	double Nzx = fzx / fGradMag - fz*(fxx*fx + fxy*fy + fzx*fz) / pow(fGradMag,3) ; 

	double Nxy = fxy / fGradMag - fx*(fxy*fx + fyy*fy + fyz*fz) / pow(fGradMag,3) ;
	double Nyy = fyy / fGradMag - fy*(fxy*fx + fyy*fy + fyz*fz) / pow(fGradMag,3) ;
	double Nzy = fyz / fGradMag - fz*(fxy*fx + fyy*fy + fyz*fz) / pow(fGradMag,3) ;

	double Nxz = fzx / fGradMag - fx*(fzx*fx + fyz*fy + fzz*fz) / pow(fGradMag,3) ;
	double Nyz = fyz / fGradMag - fy*(fzx*fx + fyz*fy + fzz*fz) / pow(fGradMag,3) ;
	double Nzz = fzz / fGradMag - fz*(fzx*fx + fyz*fy + fzz*fz) / pow(fGradMag,3) ;

	// NormalCurvature.
	vx = Nxx * tx + Nxy * ty + Nxz * tz; 
	vy = Nyx * tx + Nyy * ty + Nyz * tz; 
	vz = Nzx * tx + Nzy * ty + Nzz * tz; 

	NormalCurvature[ind] = - (tx*vx + ty*vy + tz*vz);

	// GeodesicTorsion, BPerpendicular
	vx = Nxx * nx + Nxy * ny + Nxz * nz; 
	vy = Nyx * nx + Nyy * ny + Nyz * nz; 
	vz = Nzx * nx + Nzy * ny + Nzz * nz; 

	GeodesicTorsion[ind] = - (tx*vx + ty*vy + tz*vz);
	BPerpendicular[ind] = - (nx*vx + ny*vy + nz*vz);

	/*primal of Heaviside(A), Heaviside(A), DiracDelta(A)*/

	// calculate Heaviside function
	double px = (AHPrimal[right] - AHPrimal[left]) / (2*dx);
	double py = (AHPrimal[front] - AHPrimal[back]) / (2*dy);
	double pz = (AHPrimal[up] - AHPrimal[down]) / (2*dz);

	double dot_DAHPrimal_DF = px*ax + py*ay + pz*az;

	AHeaviside[ind] = dot_DAHPrimal_DF / pow(aGradMag,2);

	// calculate DiraDelta function
	double pxx = (AHPrimal[right] - 2*AHPrimal[ind] +AHPrimal[left]) / (dx*dx);
	double pyy = (AHPrimal[front] - 2*AHPrimal[ind] + AHPrimal[back]) / (dy*dy);
	double pzz = (AHPrimal[up] - 2*AHPrimal[ind] + AHPrimal[down]) / (dz*dz);
	double pLaplacian = pxx + pyy + pzz;

	ADiracDelta[ind] = pLaplacian/pow(aGradMag,2) - dot_DAHPrimal_DF*aLaplacian/pow(aGradMag,4);

}






















